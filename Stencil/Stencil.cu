/*********************************************************************************
  *FileName:  Stencil
  *Author:  Glinttsd
  *Version:  1.0
  *Date:  2020.10.23
  *Description: 本代码利用GPU并行加速，对数组进行一个类似卷积的操作，将数组某一元素的
  *			 左右RADIUS范围内的所有元素相加，存储到另一数组里
  *Others:  数组长度N不宜太大，本示例的目的是强调share memory的合理使用。在线程过多的
  *			情况下应该分多个block进行计算，本示例只用了一个block。
**********************************************************************************/
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <malloc.h>

#define  N 16 // N不应太大(1-255)否则一个block中会有太多线程
#define  RADIUS 3

void init_vec(int* a)
{
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
	}
}

void func_print(int* b)
{
	for (int i = 0; i < N; i++)
		printf("%d\n", b[i]);
}

__global__ void stencil_kernel(int *in, int *out)
{
	int ID_local = threadIdx.x; //线程的本地坐标
	int ID_global = blockIdx.x * blockDim.x + threadIdx.x; //线程的全局坐标
	
	__shared__ int share_in[N + 2 * RADIUS];//申请share memory(SM)
	
	//初始化SM
	if (ID_local < RADIUS)
	{
		share_in[ID_local] = 0;
		share_in[(N + 2 * RADIUS) - ID_local] = 0;
	}
	share_in[ID_local + RADIUS] = in[ID_global];

	__syncthreads();//同步块中的每个线程，避免冲突
	
	//对数据进行操作
	int value = 0;
	for (int offset = -RADIUS; offset < RADIUS + 1; offset++)
	{
		value += share_in[ID_local + RADIUS + offset];
	}
	out[ID_global] = value;
}


int main()
{
	//在CPU申请内存空间
	int* a = (int*)malloc(sizeof(int) * N);
	int* b = (int*)malloc(sizeof(int) * N);

	//在GPU申请内存空间
	int* dev_a, *dev_b;
	hipMalloc((void**)&dev_a, sizeof(int) * N);
	hipMalloc((void**)&dev_b, sizeof(int) * N);

	//初始化数组
	init_vec(a);

	//将CPU中数据拷贝到GPU内存中
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);

	//加载kernel函数
	stencil_kernel <<<1, N>>> (dev_a, dev_b);// 只用到一个block

	//将GPU数据拷贝到CPU中
	hipMemcpy(b, dev_b, N * sizeof(int), hipMemcpyDeviceToHost);

	//释放GPU内存
	hipFree(dev_a);
	hipFree(dev_b);

	//输出结果
	func_print(b);
	return 0;
}