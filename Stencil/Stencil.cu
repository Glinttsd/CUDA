/*********************************************************************************
  *FileName:  Stencil
  *Author:  Glinttsd
  *Version:  1.0
  *Date:  2020.10.23
  *Description: ����������GPU���м��٣����������һ�����ƾ���Ĳ�����������ĳһԪ�ص�
  *			 ����RADIUS��Χ�ڵ�����Ԫ����ӣ��洢����һ������
  *Others:  ���鳤��N����̫�󣬱�ʾ����Ŀ����ǿ��share memory�ĺ���ʹ�á����̹߳����
  *			�����Ӧ�÷ֶ��block���м��㣬��ʾ��ֻ����һ��block��
**********************************************************************************/
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <malloc.h>

#define  N 16 // N��Ӧ̫��(1-255)����һ��block�л���̫���߳�
#define  RADIUS 3

void init_vec(int* a)
{
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
	}
}

void func_print(int* b)
{
	for (int i = 0; i < N; i++)
		printf("%d\n", b[i]);
}

__global__ void stencil_kernel(int *in, int *out)
{
	int ID_local = threadIdx.x; //�̵߳ı�������
	int ID_global = blockIdx.x * blockDim.x + threadIdx.x; //�̵߳�ȫ������
	
	__shared__ int share_in[N + 2 * RADIUS];//����share memory(SM)
	
	//��ʼ��SM
	if (ID_local < RADIUS)
	{
		share_in[ID_local] = 0;
		share_in[(N + 2 * RADIUS) - ID_local] = 0;
	}
	share_in[ID_local + RADIUS] = in[ID_global];

	__syncthreads();//ͬ�����е�ÿ���̣߳������ͻ
	
	//�����ݽ��в���
	int value = 0;
	for (int offset = -RADIUS; offset < RADIUS + 1; offset++)
	{
		value += share_in[ID_local + RADIUS + offset];
	}
	out[ID_global] = value;
}


int main()
{
	//��CPU�����ڴ�ռ�
	int* a = (int*)malloc(sizeof(int) * N);
	int* b = (int*)malloc(sizeof(int) * N);

	//��GPU�����ڴ�ռ�
	int* dev_a, *dev_b;
	hipMalloc((void**)&dev_a, sizeof(int) * N);
	hipMalloc((void**)&dev_b, sizeof(int) * N);

	//��ʼ������
	init_vec(a);

	//��CPU�����ݿ�����GPU�ڴ���
	hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);

	//����kernel����
	stencil_kernel <<<1, N>>> (dev_a, dev_b);// ֻ�õ�һ��block

	//��GPU���ݿ�����CPU��
	hipMemcpy(b, dev_b, N * sizeof(int), hipMemcpyDeviceToHost);

	//�ͷ�GPU�ڴ�
	hipFree(dev_a);
	hipFree(dev_b);

	//������
	func_print(b);
	return 0;
}